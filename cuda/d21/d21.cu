#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

const int N = 1000;
const int MAX_LINE_LENGTH = 11;

/* a utility function to calculate the hash of part of a string */
/* given the inputs str: "abcdefg", start: 3, end: 6 */
/* this function will return the hash for "def" which is 193489332 */
__device__ unsigned int djb2_hash(const char *str, int start, int end) {
    unsigned int hash = 5381;
    int c;
    int i = start;
    str += start;
    while (i < end && (c = *str++)) {
        hash = ((hash << 5) + hash) + c; /* hash * 33 + c */
        i++;
    }
    return hash;
}

__global__ void djb2_hash_kernel(const char *input, int *inputLens, unsigned int *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // this function should call djb2_hash() to calculate the hash for each input
    // remember, you do NOT need a loop here, a SINGLE LINE is sufficient
    // this is a SIMT situation, do not use a loop
    output[idx] = djb2_hash(input, 0, inputLens[idx]);
}


int main()
{    
    /*
     Your strategy here should be to:
     1) create memory space to put the text on the host
     2) read the file into that memory space, with each line as a string (input)
     3) store the length of each string in a separate int array (inputLens)
     4) put the strings into one giant string
     5) copy the "giant string" to the GPU
     6) run the djb2_hash_kernel
     7) get the output from the device back into the host (h_output)
     8) print the output to the screen (the for loop I provide below)
     */
    char lines[N][MAX_LINE_LENGTH];
    char *input = (char *)malloc(N * MAX_LINE_LENGTH * sizeof(char));
    int *inputLens = (int *)malloc(N * sizeof(int));
    
    FILE *file = fopen("random_strings.txt", "r");
    
    char buffer[MAX_LINE_LENGTH + 1];
    for (int i = 0; i < N; i++) {
        if (fgets(buffer, sizeof(buffer), file) != NULL) {
            int len = strlen(buffer);
            if (buffer[len - 1] == '\n') {
                buffer[len - 1] = '\0';
            }
            strcpy(input + (i * MAX_LINE_LENGTH), buffer);
            strcpy(lines[i], buffer);
            inputLens[i] = len;
        }
    }
    fclose(file);
    
    char *d_input;
    int *d_inputLens;
    unsigned int *d_output;
    hipMalloc((void **)&d_input, N * MAX_LINE_LENGTH * sizeof(char));
    hipMalloc((void **)&d_inputLens, N * sizeof(int));
    hipMalloc((void **)&d_output, N * sizeof(unsigned int));
    hipMemcpy(d_input, input, N * MAX_LINE_LENGTH * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_inputLens, inputLens, N * sizeof(int), hipMemcpyHostToDevice);
    
    djb2_hash_kernel<<<8, 125>>>(d_input, d_inputLens, d_output);

    unsigned int *output = (unsigned int *)malloc(N * sizeof(unsigned int));
    hipMemcpy(output, d_output, N * sizeof(unsigned int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
    {
        printf("%d: string %s hash %u\n", i, lines[i], output[i]);
    }
    
    free(input);
    free(inputLens);
    free(output);
    hipFree(d_input);
    hipFree(d_inputLens);
    hipFree(d_output);

    return 0;
}
